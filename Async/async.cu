#include <hip/hip_runtime.h>

#include <stdio.h>

#define N 1000000

__global__ void gpuAdd(int *d_a, int *d_b, int *d_c){
    //总线程id = 当前块线程id.x + 块id*块维度x
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		d_c[tid] = d_a[tid] + d_b[tid];// 加法
		tid += blockDim.x * gridDim.x; // 一次执行一个格子 块维度x*格子维度x
	}
}

int main(){
    int *h_a, *h_b, *h_c;
    int *d_a0, *d_b0, *d_c0;
    int *d_a1, *d_b1, *d_c1;
    int sizeByte = N*sizeof(int)*2;

    //create two streams
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

    //Use cudaHostMalloc to allocate page-locked memory
    hipHostAlloc(&h_a, sizeByte, hipHostMallocDefault);
    hipHostAlloc(&h_b, sizeByte, hipHostMallocDefault);
    hipHostAlloc(&h_c, sizeByte, hipHostMallocDefault);
    //same as single stream, just add stream parameter when launching kernel
    for(int i=0;i<N*2;i++){
        h_a[i] = i;
        h_b[i] = i;
    }

    hipMalloc(&d_a0,sizeByte/2);
    hipMalloc(&d_b0,sizeByte/2);
    hipMalloc(&d_c0,sizeByte/2);
    hipMalloc(&d_a1,sizeByte/2);
    hipMalloc(&d_b1,sizeByte/2);
    hipMalloc(&d_c1,sizeByte/2);

    hipMemcpyAsync(d_a0, h_a, sizeByte/2, hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(d_a1, h_a + N, sizeByte/2, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_b0, h_b, sizeByte/2, hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(d_b1, h_b + N, sizeByte/2, hipMemcpyHostToDevice, stream1);

    gpuAdd<<<512,512,0,stream0>>>(d_a0,d_b0,d_c0);
    gpuAdd<<<512,512,0,stream1>>>(d_a1,d_b1,d_c1);

    hipMemcpyAsync(h_c, d_c0, sizeByte/2, hipMemcpyDeviceToHost, stream0);
    hipMemcpyAsync(h_c+N, d_c1, sizeByte/2, hipMemcpyDeviceToHost, stream1);

    //only synchronize on cpu/host
    hipDeviceSynchronize();
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);


    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    printf("Time consumption: %lf\n", time);

    hipEventDestroy(stop);
    hipEventDestroy(start);

    int Correct = 1;
    int wrongIndex = -1;
	printf("Vector addition on GPU \n");
	//Printing result on console
	for (int i = 0; i < 2*N; i++) 
	{
		if ((h_a[i] + h_b[i] != h_c[i]))
		{
            Correct = 0;
            wrongIndex = i;
            break;
		}

	}
	if (Correct == 1)
	{
		printf("GPU has computed Sum Correctly\n");
	}
	else
	{
        printf("There is an Error in GPU Computation, at index %d, CPU(%d)!=GPU(%d)\n", wrongIndex, (h_a[wrongIndex] + h_b[wrongIndex]), h_c[wrongIndex]);
	}

    // 清空GPU内存
	hipFree(d_a0);
	hipFree(d_b0);
	hipFree(d_c0);
	hipFree(d_a0);
	hipFree(d_b0);
	hipFree(d_c0);
	
	// 清空cuda分配的cpu内存
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);
	return 0;
}